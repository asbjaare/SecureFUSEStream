#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <libavcodec/avcodec.h>
#include <libavformat/avformat.h>
#include <libswscale/swscale.h>
#include <stdio.h>

// CUDA kernel to process frame data
__global__ void process_frame(uint8_t *data, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    int idx = y * width + x;
    data[idx] = 255 - data[idx]; // Example: Invert pixel values
  }
}

int main(int argc, char *argv[]) {
  av_register_all();

  if (argc < 2) {
    printf("Usage: %s <video file>\n", argv[0]);
    return -1;
  }

  AVFormatContext *pFormatContext = avformat_alloc_context();
  if (!pFormatContext) {
    printf("ERROR: Could not allocate memory for Format Context\n");
    return -1;
  }

  if (avformat_open_input(&pFormatContext, argv[1], NULL, NULL) != 0) {
    printf("ERROR: Could not open the file\n");
    return -1;
  }

  if (avformat_find_stream_info(pFormatContext, NULL) < 0) {
    printf("ERROR: Could not get the stream info\n");
    return -1;
  }

  AVCodec *pCodec = NULL;
  AVCodecParameters *pCodecParameters = NULL;
  int video_stream_index = -1;

  for (int i = 0; i < pFormatContext->nb_streams; i++) {
    AVCodecParameters *pLocalCodecParameters =
        pFormatContext->streams[i]->codecpar;
    AVCodec *pLocalCodec =
        avcodec_find_decoder(pLocalCodecParameters->codec_id);

    if (pLocalCodecParameters->codec_type == AVMEDIA_TYPE_VIDEO) {
      video_stream_index = i;
      pCodec = pLocalCodec;
      pCodecParameters = pLocalCodecParameters;

      printf("Video Codec: resolution %d x %d\n", pLocalCodecParameters->width,
             pLocalCodecParameters->height);
      break;
    }
  }

  if (video_stream_index == -1) {
    printf("ERROR: Could not find a video stream in the file\n");
    return -1;
  }

  AVCodecContext *pCodecContext = avcodec_alloc_context3(pCodec);
  if (!pCodecContext) {
    printf("ERROR: failed to allocated memory for AVCodecContext\n");
    return -1;
  }

  if (avcodec_parameters_to_context(pCodecContext, pCodecParameters) < 0) {
    printf("ERROR: failed to copy codec params to codec context\n");
    return -1;
  }

  if (avcodec_open2(pCodecContext, pCodec, NULL) < 0) {
    printf("ERROR: failed to open codec through avcodec_open2\n");
    return -1;
  }

  AVFrame *pFrame = av_frame_alloc();
  AVPacket *pPacket = av_packet_alloc();

  while (av_read_frame(pFormatContext, pPacket) >= 0) {
    if (pPacket->stream_index == video_stream_index) {
      int response = avcodec_send_packet(pCodecContext, pPacket);
      if (response < 0) {
        printf("ERROR: Failed to decode packet\n");
        continue;
      }

      response = avcodec_receive_frame(pCodecContext, pFrame);
      if (response == AVERROR(EAGAIN) || response == AVERROR_EOF) {
        continue;
      } else if (response < 0) {
        printf("ERROR: Failed to receive frame\n");
        return -1;
      }

      // Allocate device memory and copy frame data
      uint8_t *d_frame;
      int frame_size = pFrame->width * pFrame->height;
      hipMalloc((void **)&d_frame, frame_size);
      hipMemcpy(d_frame, pFrame->data[0], frame_size, hipMemcpyHostToDevice);

      // Define block and grid sizes
      dim3 threadsPerBlock(16, 16);
      dim3 numBlocks(
          (pFrame->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
          (pFrame->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

      // Launch CUDA kernel
      process_frame<<<numBlocks, threadsPerBlock>>>(d_frame, pFrame->width,
                                                    pFrame->height);
      hipDeviceSynchronize();

      // Copy processed data back to host
      hipMemcpy(pFrame->data[0], d_frame, frame_size, hipMemcpyDeviceToHost);

      // Here you can save or display the processed frame
      printf("Processed Frame %d (type=%c, size=%d bytes) pts %ld key_frame %d "
             "[DTS %d]\n",
             pCodecContext->frame_number,
             av_get_picture_type_char(pFrame->pict_type), pFrame->pkt_size,
             pFrame->pts, pFrame->key_frame, pFrame->coded_picture_number);

      // Free device memory
      hipFree(d_frame);
    }
    av_packet_unref(pPacket);
  }

  av_frame_free(&pFrame);
  av_packet_free(&pPacket);
  avcodec_free_context(&pCodecContext);
  avformat_close_input(&pFormatContext);
  avformat_free_context(pFormatContext);

  return 0;
}
